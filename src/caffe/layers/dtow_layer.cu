#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/dtow_layer.hpp"

namespace caffe {
	template <typename Dtype>
	__global__ void dtow_kernel(const int nthreads, const Dtype* const bottom_data,
		const int num, const int channels, const int height, const int width,
		const int channels_out, const int height_out, const int width_out, const int patch_size,
		Dtype* const top_data) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int tw = index%width;
			int th = (index / width) % height;
			int tc = (index / width / height) % channels;
			int	tn = index / width / height / channels;
			int p2size = patch_size*patch_size;
			int pc = tc / p2size;
			int rc = tc % p2size;
			int ph = th*patch_size + rc / patch_size;
			int pw = tw*patch_size + rc % patch_size;
			int pidx = ((tn*channels_out + pc)*height_out + ph)*width_out + pw;
			top_data[pidx] = bottom_data[index];

		}
	}
	template <typename Dtype>
	__global__ void wtod_kernel(const int nthreads, const Dtype* const bottom_data,
		const int num, const int channels, const int height, const int width,
		const int channels_out, const int height_out, const int width_out, const int patch_size,
		Dtype* const top_data) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int tw = index%width;
			int th = (index / width) % height;
			int tc = (index / width / height) % channels;
			int	tn = index / width / height / channels;
			int p2size = patch_size*patch_size;
			int ph = th / patch_size;
			int pw = tw / patch_size;
			int pc = tc * p2size + (th%patch_size)*patch_size + tw%patch_size;
			int pidx = ((tn*channels_out + pc)*height_out + ph)*width_out + pw;
			top_data[pidx] = bottom_data[index];

		}
	}
	template <typename Dtype>
	void DtowLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		Dtype* const top_data = top[0]->mutable_gpu_data();
		const Dtype* bottom_data = bottom[0]->gpu_data();
		int count = bottom[0]->count();
		if (d2w){
			dtow_kernel<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
				(count, bottom_data, num_, ch_in, h_in, w_in, ch_out, h_out, w_out, psize, top_data);
		}
		else{
			wtod_kernel<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
				(count, bottom_data, num_, ch_in, h_in, w_in, ch_out, h_out, w_out, psize, top_data);
		}
		
		CUDA_POST_KERNEL_CHECK;
	}
	template <typename Dtype>
	__global__ void dtow_backward_kernel(const int nthreads, const Dtype* const top_diff,
		const int num, const int channels, const int height, const int width,
		const int channels_out, const int height_out, const int width_out, const int patch_size,
		Dtype* const bottom_diff) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int tw = index%width;
			int th = (index / width) % height;
			int tc = (index / width / height) % channels;
			int	tn = index / width / height / channels;
			int p2size = patch_size*patch_size;
			int pc = tc / p2size;
			int rc = tc % p2size;
			int ph = th*patch_size + rc / patch_size;
			int pw = tw*patch_size + rc % patch_size;
			int pidx = ((tn*channels_out + pc)*height_out + ph)*width_out + pw;
			bottom_diff[index] = top_diff[pidx];
		}
	}
	template <typename Dtype>
	__global__ void wtod_backward_kernel(const int nthreads, const Dtype* const top_diff,
		const int num, const int channels, const int height, const int width,
		const int channels_out, const int height_out, const int width_out, const int patch_size,
		Dtype* const bottom_diff) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int tw = index%width;
			int th = (index / width) % height;
			int tc = (index / width / height) % channels;
			int	tn = index / width / height / channels;
			int p2size = patch_size*patch_size;
			int ph = th / patch_size;
			int pw = tw / patch_size;
			int pc = tc * p2size + (th%patch_size)*patch_size + tw%patch_size;
			int pidx = ((tn*channels_out + pc)*height_out + ph)*width_out + pw;
			bottom_diff[index] = top_diff[pidx];
		}
	}
	template <typename Dtype>
	void DtowLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		const Dtype* const top_diff = top[0]->gpu_diff();
		Dtype* const bottom_diff = bottom[0]->mutable_gpu_diff();
		int count = bottom[0]->count();
		if (d2w){
			dtow_backward_kernel<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
				(count, top_diff, num_, ch_in, h_in, w_in, ch_out, h_out, w_out, psize, bottom_diff);
		}
		else{
			wtod_backward_kernel<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
				(count, top_diff, num_, ch_in, h_in, w_in, ch_out, h_out, w_out, psize, bottom_diff);
		}
		
		//LOG(INFO) << "1";
		CUDA_POST_KERNEL_CHECK;
	}

	INSTANTIATE_LAYER_GPU_FUNCS(DtowLayer);

}  // namespace caffe
